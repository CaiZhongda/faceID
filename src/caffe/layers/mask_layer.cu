#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/mask_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaskCopyGPU(const int count, const int num, const int channels,
			      const Dtype*  bottom_data, Dtype* mask){
	const int spatial_dim = count / num / channels;
	const int dim = spatial_dim * channels;
	CUDA_KERNEL_LOOP(index, count){
		const int bottom_idx = (index / dim) * spatial_dim
					+ (index % spatial_dim);
		mask[index] = bottom_data[bottom_idx];
	}
}

template <typename Dtype>
void MaskLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int count = top[0]->count();
  const int num = bottom[0]->num();
  const int channels = bottom[0]->channels();
  const int height = bottom[0]->height();
  const int width = bottom[0]->width();
  
  if(0){ //DEBUG
    for(int i = 0; i < height * width; i++)
	LOG(INFO) << "mask2d[" << i << "] = " << bottom[1]->cpu_data()[i];
  }
  
  // use the bottom[1] Nx1xHxW mask to create the NxCxHxW mask, by duplicate each HxW map for C times.
  MaskCopyGPU<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, 
			num, channels, bottom[1]->gpu_data(), mask.mutable_gpu_data());
  // scale [default = 1.0]
  caffe_gpu_scale(count, Dtype(scale), mask.gpu_data(), mask.mutable_gpu_data());

  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_mul(count, bottom[0]->gpu_data(), mask.gpu_data(), top_data);
   
  if(0){ //DEBUG
    for(int i = width * 6; i < width * 7; i++)
	if(mask.cpu_data()[i] < Dtype(0.18))
	LOG(INFO) << "mask[" << i << "] = " << mask.cpu_data()[i];
  }
}


template <typename Dtype>
__global__ void MaskBackwardGPU(const int count, const int num, const int channels,
	             const Dtype* tmp, Dtype* bottom_diff){
	const int spatial_dim = count / num / channels;
	const int dim = spatial_dim * channels;
	CUDA_KERNEL_LOOP(index, count){
		const int bottom_idx = (index / dim) * spatial_dim
					+ (index % spatial_dim);
		bottom_diff[bottom_idx] += tmp[index];
	}
}


template <typename Dtype>
void MaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const int count = top[0]->count();
  const int num = bottom[0]->num();
  const int channels = bottom[0]->channels();
  const int height = bottom[0]->height();
  const int width = bottom[0]->width();
  const Dtype* top_diff = top[0]->gpu_diff();
  for (int i = 0; i < bottom.size(); ++i) {
    if (propagate_down[i]) {
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
	if(i == 0){ // diff for bottom feature map
		caffe_gpu_mul(count, mask.gpu_data(), top_diff, bottom_diff);
	} else{ // diff for bottom mask
		caffe_gpu_set(bottom[i]->count(), Dtype(0), bottom_diff);
		// multiply top_diff and feature maps element-wisely. (1)
		caffe_gpu_mul(count, bottom[0]->gpu_data(), top_diff, tmp.mutable_gpu_data());
		// scale [defaut = 1.0]
		caffe_gpu_scale(count, Dtype(scale), tmp.gpu_data(), tmp.mutable_gpu_data());
		// accumulate the multiplication values in (1) along the channel dimension
		MaskBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, 
				num, channels, tmp.gpu_data(), bottom_diff);
		if(0){  //DEBUG
			for(int k = 0; k < height * width / 13; k++){
				LOG(INFO) << "mask_diff[" << k << "]=" << bottom[1]->cpu_diff()[k];	
				//LOG(INFO) << "tmp[" << k << "]=" << tmp.cpu_data()[k];
				//LOG(INFO) << "tmp[" << k + height * width << "]=" << tmp.cpu_data()[k+height*width];
			}
		}
		
	}//if(i == 0)
    }//if(propagate_down[0])
  }//for
}

INSTANTIATE_LAYER_GPU_FUNCS(MaskLayer);

}  // namespace caffe
