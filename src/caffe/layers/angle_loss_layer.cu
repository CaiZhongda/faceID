#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/angle_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Compute_loss_gpu(const int M, const Dtype* dotxb, const Dtype* dotx, const Dtype* dotb, 
				 Dtype* loss) {
  CUDA_KERNEL_LOOP(index, M) {
    loss[index] = Dtype(1.0) - dotxb[index] * dotxb[index] / (dotx[index] * dotb[index] + Dtype(1e-8)); 
  }
}

template <typename Dtype>
__global__ void Compute_center_diff_gpu(int nthreads, const int M, const int K, const Dtype* x,
        const Dtype* b, const Dtype* label, const Dtype* dotxb, const Dtype* dotx, const Dtype* dotb,
        Dtype* center_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int count = 0;
    for (int m = 0; m < M; m++) {
      const int label_value = static_cast<int>(label[m]);
      if (label_value == index) {
        count++;
	Dtype D = dotx[m];
        for (int k = 0; k < K; k++) {
          int idx_x = m * K + k;
	  int idx_b = index * K + k;
    	  Dtype x_val = x[idx_x];
          Dtype b_val = b[idx_b];
    	  Dtype cross_term = dotxb[m] - x_val * b_val;
    	  Dtype A = x_val * x_val;
    	  Dtype B = 2 * x_val * cross_term;
    	  Dtype C = cross_term * cross_term;
    	  Dtype E = dotx[m] * (dotb[m] - b_val * b_val);
          center_diff[idx_b] += Dtype(-1.0) * ((Dtype(-1.0)*D*B*b_val*b_val + 2*(A*E - C*D)*b_val + B*E)
				                    / (dotx[m] * dotx[m] * dotb[m] * dotb[m]  + 1e-8));
        }
      }
    }
    if(count > 1){
    	for (int k = 0; k < K; k++) {
     	  center_diff[index * K + k] = center_diff[index * K + k] / Dtype(count);
    	}
    }
  }
}

template <typename Dtype>
__global__ void Compute_bottom_diff_gpu(int nthreads, int K, const Dtype* x,
        const Dtype* b, const Dtype* label, const Dtype* dotxb, const Dtype* dotx, const Dtype* dotb,
        Dtype* bottom_diff){
  CUDA_KERNEL_LOOP(index, nthreads){
    int m = index / K;
    int k = index % K;
    int label_value = static_cast<int>(label[m]);
    Dtype x_val = x[index];
    Dtype b_val = b[label_value * K + k];
    Dtype cross_term = dotxb[m] - x_val * b_val;
    Dtype A = b_val * b_val;
    Dtype B = 2 * b_val * cross_term;
    Dtype C = cross_term * cross_term;
    Dtype D = dotb[m];
    Dtype E = dotb[m] * (dotx[m] - x_val * x_val);
    bottom_diff[index] = Dtype(-1.0) * ((Dtype(-1.0)*D*B*x_val*x_val + 2*(A*E - C*D)*x_val + B*E)
                            / (dotx[m] * dotx[m] * dotb[m] * dotb[m] + 1e-8));
  }
}


template <typename Dtype>
void AngleLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* x = bottom[0]->gpu_data();
  const Dtype* b = this->blobs_[0]->gpu_data();
  Dtype mean_x_sq = 0;
  for(int i = 0; i < M_; i++){
	const int label_value = static_cast<int>(bottom[1]->cpu_data()[i]);
	const Dtype* px = &x[i * K_];
	const Dtype* pb = &b[label_value * K_];
	Dtype tmp;
	caffe_gpu_dot(K_, px, pb, &tmp);
	if(0){
		LOG(INFO) << "dotxb[" << i << "]=" << tmp;
	}
	dotxb.mutable_cpu_data()[i] = tmp;
	caffe_gpu_dot(K_, px, px, &tmp);
	if(1){
		//LOG(INFO) << "dotx = " << tmp;
		mean_x_sq += tmp;
	}
	dotx.mutable_cpu_data()[i] = tmp;
	caffe_gpu_dot(K_, pb, pb, &tmp);
	dotb.mutable_cpu_data()[i] = tmp;
	if(0){
		LOG(INFO) << "dotb=" << tmp;
		LOG(INFO) << " ";
	}
  }
  if(1){
	LOG(INFO) << "mean of dotx=" << mean_x_sq / Dtype(M_);
  }
  Compute_loss_gpu<Dtype><<<CAFFE_GET_BLOCKS(M_),
      CAFFE_CUDA_NUM_THREADS>>>(M_, dotxb.gpu_data(), dotx.gpu_data(), 
				dotb.gpu_data(), loss_data.mutable_gpu_data());
  Dtype loss = 0.0;
  caffe_gpu_asum(M_, loss_data.gpu_data(), &loss);
  loss = loss / M_;
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void AngleLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  int nthreads = N_;
  const Dtype* x = bottom[0]->gpu_data();
  const Dtype* b = this->blobs_[0]->gpu_data();
  Dtype* blob_diff = this->blobs_[0]->mutable_gpu_diff();
  caffe_gpu_set(N_ * K_, (Dtype)0., blob_diff);
  // diff for center parameters (i.e. b)
  Compute_center_diff_gpu<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, M_, K_, x, b, bottom[1]->gpu_data(), dotxb.gpu_data(), dotx.gpu_data(),
                                dotb.gpu_data(), blob_diff);
  caffe_gpu_scal(N_ * K_, Dtype(0.1), blob_diff);
  if (propagate_down[0]) {
  // diff for bottom[0]
    Compute_bottom_diff_gpu<Dtype><<<CAFFE_GET_BLOCKS(M_ * K_),
        CAFFE_CUDA_NUM_THREADS>>>(M_ * K_, K_, x, b, bottom[1]->gpu_data(), dotxb.gpu_data(), dotx.gpu_data(),
                                  dotb.gpu_data(), bottom[0]->mutable_gpu_diff());
    caffe_gpu_scal(M_ * K_, top[0]->cpu_diff()[0] / M_, bottom[0]->mutable_gpu_diff());
    if(1){ //DEBUG
	for(int i = 0; i < 4; i++){
		LOG(INFO) << "bottom_diff[" << i << "]=" << bottom[0]->cpu_diff()[i];
	}
	int label_value = bottom[1]->cpu_data()[0];
	for(int i = 0; i < 4; i++){
		LOG(INFO) << "center_diff[" << i << "]=" << this->blobs_[0]->cpu_diff()[label_value * K_ + i];
	}
    }
  }
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AngleLossLayer);

}  // namespace caffe
